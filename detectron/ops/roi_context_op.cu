#include "hip/hip_runtime.h"
#include <cfloat>

#include "caffe2/core/context_gpu.h"
#include "roi_context_op.h"

namespace caffe2 {

namespace {

template <typename T>
__global__ void ROIContextForward(const int nthreads, const T* rois,
                                  const float context_ratio_, const int max_h,
                                  const int max_w, T* Frois, T* Crois) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int n = index;

    const T* offset_rois = rois + n * 5;
    T* offset_Frois = Frois + n * 9;
    T* offset_Crois = Crois + n * 9;

    // x1 y1 x2 y2
    float x1 = offset_rois[1];
    float y1 = offset_rois[2];
    float x2 = offset_rois[3];
    float y2 = offset_rois[4];

    float rois_w = x2 - x1;
    float rois_h = y2 - y1;

    float rois_inner_w = rois_w / context_ratio_;
    float rois_inner_h = rois_h / context_ratio_;

    float rois_outer_w = rois_w * context_ratio_;
    float rois_outer_h = rois_h * context_ratio_;

    float inner_residual_w = rois_w - rois_inner_w;
    float inner_residual_h = rois_h - rois_inner_h;

    float outer_residual_w = rois_outer_w - rois_w;
    float outer_residual_h = rois_outer_h - rois_h;

    offset_Frois[0] = offset_rois[0];
    offset_Frois[1] = offset_rois[1];
    offset_Frois[2] = offset_rois[2];
    offset_Frois[3] = offset_rois[3];
    offset_Frois[4] = offset_rois[4];
    offset_Frois[5] = offset_rois[1];
    offset_Frois[6] = offset_rois[2];
    offset_Frois[7] = offset_rois[3];
    offset_Frois[8] = offset_rois[4];

    offset_Crois[0] = offset_rois[0];
    offset_Crois[1] = offset_rois[1];
    offset_Crois[2] = offset_rois[2];
    offset_Crois[3] = offset_rois[3];
    offset_Crois[4] = offset_rois[4];
    offset_Crois[5] = offset_rois[1];
    offset_Crois[6] = offset_rois[2];
    offset_Crois[7] = offset_rois[3];
    offset_Crois[8] = offset_rois[4];

    offset_Frois[5] += inner_residual_w / 2;
    offset_Frois[6] += inner_residual_h / 2;
    offset_Frois[7] -= inner_residual_w / 2;
    offset_Frois[8] -= inner_residual_h / 2;

    offset_Crois[1] -= outer_residual_w / 2;
    offset_Crois[2] -= outer_residual_h / 2;
    offset_Crois[3] += outer_residual_w / 2;
    offset_Crois[4] += outer_residual_h / 2;

    offset_Frois[5] = min(max(offset_Frois[5], T(0)), T(max_w));
    offset_Frois[6] = min(max(offset_Frois[6], T(0)), T(max_h));
    offset_Frois[7] = min(max(offset_Frois[7], T(0)), T(max_w));
    offset_Frois[8] = min(max(offset_Frois[8], T(0)), T(max_h));

    offset_Crois[1] = min(max(offset_Crois[1], T(0)), T(max_w));
    offset_Crois[2] = min(max(offset_Crois[2], T(0)), T(max_h));
    offset_Crois[3] = min(max(offset_Crois[3], T(0)), T(max_w));
    offset_Crois[4] = min(max(offset_Crois[4], T(0)), T(max_h));
  }
}

}  // namespace

template <>
bool RoIContextOp<float, HIPContext>::RunOnDevice() {
  auto& R = Input(0);
  auto& X = Input(1);
  auto* RF = Output(0);
  auto* RC = Output(1);

  const int num_rois = R.dim32(0);
  const int num_channels = R.dim32(1);

  CAFFE_ENFORCE_EQ(num_channels, 5);

  RF->Resize(num_rois, 9);
  RC->Resize(num_rois, 9);

  ROIContextForward<float><<<CAFFE_GET_BLOCKS(num_rois), CAFFE_CUDA_NUM_THREADS,
                             0, context_.hip_stream()>>>(
      num_rois, R.data<float>(), context_ratio_, X.dim32(2), X.dim32(3),
      RF->template mutable_data<float>(), RC->template mutable_data<float>());
  return true;
}

REGISTER_HIP_OPERATOR(RoIContext, RoIContextOp<float, HIPContext>);

}  // namespace caffe2
