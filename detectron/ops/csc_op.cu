#include "hip/hip_runtime.h"
#include <cfloat>
#include <functional>

#include "caffe2/core/context_gpu.h"
#include "csc_op.h"

namespace caffe2 {

namespace {

template <typename T>
__global__ void kernel_show(const T* Xdata, const int batch_size,
                            const int channels, const int height,
                            const int width, const int ndim, const int gpu_id,
                            const int uuid) {
  printf("uuid=%d gpu=%d ndim=%d b = %d c = %d h = %d w = %d\n", uuid, gpu_id,
         ndim, batch_size, channels, height, width);
  for (int b = 0; b < batch_size; b++) {
    for (int c = 0; c < channels; c++) {
      for (int h = 0; h < height; h++) {
        for (int w = 0; w < width; w++) {
          int index_X = ((b * channels + c) * height + h) * width + w;
          printf("b = %d c = %d h = %d w = %d %.32f\n", b, c, h, w,
                 Xdata[index_X]);
        }
      }
    }
  }
}

template <typename T>
__global__ void kernel_show_c(const T* Xdata, const int batch_size,
                              const int channels, const int height,
                              const int width, const int ndim, const int gpu_id,
                              const int uuid, const int c) {
  printf("uuid=%d gpu=%d ndim=%d b = %d c = %d h = %d w = %d\n", uuid, gpu_id,
         ndim, batch_size, channels, height, width);
  for (int b = 0; b < batch_size; b++) {
    // for (int c = 0; c < channels; c++) {
    for (int h = 0; h < height; h++) {
      for (int w = 0; w < width; w++) {
        int index_X = ((b * channels + c) * height + h) * width + w;
        printf("b = %d c = %d h = %d w = %d %.32f\n", b, c, h, w,
               Xdata[index_X]);
      }
    }
    //}
  }
}

template <typename T>
__global__ void binary_kernel(const int nthreads, const T* const x, T* const y,
                              const T threshold) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    if (x[index] >= threshold) {
      y[index] = 1;
    } else {
      y[index] = 0;
    }
  }
}

template <typename T>
void integral_cpu(const T* src, T* sum, const int height, const int width) {
  T s = 0;
  for (int x = 0; x < width; x++) {
    s += src[x];
    sum[x] = s;
  }
  src += width;
  sum += width;
  for (int y = 1; y < height; y++, src += width, sum += width) {
    s = 0;
    for (int x = 0; x < width; x++) {
      s += src[x];
      sum[x] = sum[x - width] + s;
    }
  }
}

template <typename T>
void binary_and_integral_cpu(const T* src, T* sum, const int height,
                             const int width, const T threshold) {
  T s = 0;
  for (int x = 0; x < width; x++) {
    if (src[x] >= threshold) {
      s += 1;
    } else {
      s += 0;
    }
    sum[x] = s;
  }
  src += width;
  sum += width;
  for (int y = 1; y < height; y++, src += width, sum += width) {
    s = 0;
    for (int x = 0; x < width; x++) {
      if (src[x] >= threshold) {
        s += 1;
      } else {
        s += 0;
      }
      sum[x] = sum[x - width] + s;
    }
  }
}

template <typename T>
T get_sum(const int N, const T* data) {
  T sum_val = 0;
  for (int i = 0; i < N; i++) {
    sum_val += *data;
    data += 1;
  }
  return sum_val;
}

template <typename T>
T get_max(const int N, const T* data) {
  T max_val = -FLT_MAX;
  for (int i = 0; i < N; i++) {
    if (*data > max_val) {
      max_val = *data;
    }
    data += 1;
  }
  return max_val;
}

template <typename T>
__global__ void CSCPool(const int nthreads, const T* cpg_data,
                        const int height_im, const int width_im,
                        const T* rois_data, const int num_class,
                        const int cls_id, const T min_density, const T min_mass,
                        const bool area_sqrt, const T context_scale,
                        T* const top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int rois_index = index;

    rois_data += 5 * rois_index;
    int wstart = round(rois_data[1]);
    int hstart = round(rois_data[2]);
    int wend = round(rois_data[3]);
    int hend = round(rois_data[4]);

    // if (wstart < 0) wstart = 0;
    // if (wstart >= width_im) wstart = width_im - 1;
    // if (hstart < 0) hstart = 0;
    // if (hstart >= height_im) hstart = height_im - 1;

    // Check RoI
    if (wstart >= 0 && hstart >= 0 && wstart < wend && hstart < hend &&
        wend < width_im && hend < height_im) {
    } else {
      top_data[rois_index * num_class + cls_id] = kMIN_SCORE;
      // 这里面是for循环，用return会中断后续的循环
      continue;
    }

    // caculate the inner and outer RoI coordinate
    T width_roi = wend - wstart;
    T height_roi = hend - hstart;
    // T context_scale = 1.8;
    // T context_scale = sqrtf(2.0);
    T width_roi_inner = 1.0 * width_roi / context_scale;
    T height_roi_inner = 1.0 * height_roi / context_scale;
    T width_roi_outer = 1.0 * width_roi * context_scale;
    T height_roi_outer = 1.0 * height_roi * context_scale;
    T wcenter = 1.0 * (wend + wstart) / 2.0;
    T hcenter = 1.0 * (hend + hstart) / 2.0;

    int wstart_inner = round(wcenter - width_roi_inner / 2.0);
    int hstart_inner = round(hcenter - height_roi_inner / 2.0);
    int wend_inner = round(wcenter + width_roi_inner / 2.0);
    int hend_inner = round(hcenter + height_roi_inner / 2.0);

    int wstart_outer = round(max(wcenter - width_roi_outer / 2.0, 0.0));
    int hstart_outer = round(max(hcenter - height_roi_outer / 2.0, 0.0));
    int wend_outer =
        round(min(wcenter + width_roi_outer / 2.0, width_im - 1.0));
    int hend_outer =
        round(min(hcenter + height_roi_outer / 2.0, height_im - 1.0));

    width_roi = wend - wstart + 1;
    height_roi = hend - hstart + 1;
    width_roi_inner = wend_inner - wstart_inner + 1;
    height_roi_inner = hend_inner - hstart_inner + 1;
    width_roi_outer = wend_outer - wstart_outer + 1;
    height_roi_outer = hend_outer - hstart_outer + 1;

    // a1-a2-a3+a4
    T a1, a2, a3, a4;

    // CPG sum of RoI
    a1 = cpg_data[hend * width_im + wend];
    a2 = (wstart - 1 >= 0) ? cpg_data[hend * width_im + (wstart - 1)] : 0;
    a3 = (hstart - 1 >= 0) ? cpg_data[(hstart - 1) * width_im + wend] : 0;
    a4 = (hstart - 1 >= 0 && wstart - 1 >= 0)
             ? cpg_data[(hstart - 1) * width_im + (wstart - 1)]
             : 0;
    T sum_roi = a1 - a2 - a3 + a4;

    // CPG sum of inner RoI
    a1 = cpg_data[hend_inner * width_im + wend_inner];
    a2 = (wstart_inner - 1 >= 0)
             ? cpg_data[hend_inner * width_im + (wstart_inner - 1)]
             : 0;
    a3 = (hstart_inner - 1 >= 0)
             ? cpg_data[(hstart_inner - 1) * width_im + wend_inner]
             : 0;
    a4 = (hstart_inner - 1 >= 0 && wstart_inner - 1 >= 0)
             ? cpg_data[(hstart_inner - 1) * width_im + (wstart_inner - 1)]
             : 0;
    T sum_inner = a1 - a2 - a3 + a4;

    // CPG sum of outer RoI
    a1 = cpg_data[hend_outer * width_im + wend_outer];
    a2 = (wstart_outer - 1 >= 0)
             ? cpg_data[hend_outer * width_im + (wstart_outer - 1)]
             : 0;
    a3 = (hstart_outer - 1 >= 0)
             ? cpg_data[(hstart_outer - 1) * width_im + wend_outer]
             : 0;
    a4 = (hstart_outer - 1 >= 0 && wstart_outer - 1 >= 0)
             ? cpg_data[(hstart_outer - 1) * width_im + (wstart_outer - 1)]
             : 0;
    T sum_outer = a1 - a2 - a3 + a4;

    // area size
    T area_roi = height_roi * width_roi;
    T area_inner = height_roi_inner * width_roi_inner;
    T area_outer = height_roi_outer * width_roi_outer;

    T area_frame = max(area_roi - area_inner, T(1));
    T area_context = max(area_outer - area_roi, T(1));

    //-----------------------------------------------------------------------
    T score;
    T sum_frame = sum_roi - sum_inner;
    T sum_context = sum_outer - sum_roi;

    // current best
    if (area_sqrt) {
      score = sum_frame / sqrt(area_frame) - sum_context / sqrt(area_context);
    } else {
      score = sum_frame / area_frame - sum_context / area_context;
    }

    // bad at test debug
    // T score = (sum_roi - sum_inner) - (sum_outer - sum_roi);

    // (msra 0223):
    // T score = ((sum_roi - 2.0 * (sum_outer - sum_roi)) *
    //(2.0 * (sum_roi - sum_inner) - sum_inner)) /
    // area_roi;
    // if ((sum_roi - 2.0 * (sum_outer - sum_roi)) < 0 &&
    //(2.0 * (sum_roi - sum_inner) - sum_inner) < 0) {
    // score = -1.0 * score;
    //}

    // (msra 0101): bad
    // T score = sqrt((sum_roi - sum_inner) / area_frame) -
    //               sqrt((sum_outer - sum_roi) / area_context);

    // (msra 12.30): very bad
    // T score =
    //    (sum_roi - sum_inner) / area_frame - (sum_outer - sum_roi) /
    // area_context;

    // (msra 12.29): bad
    // T score = ((sum_roi - sum_inner) - (sum_outer - sum_roi)) /
    // area_frame;

    // (msra 0105): bad than (msra 12.29)
    // T score = ((sum_roi - sum_inner) - (sum_outer - sum_roi)) /
    // sqrt(area_frame);

    //-----------------------------------------------------------------------

    // if (sum_roi < min_mass) score = kMIN_SCORE;

    top_data[rois_index * num_class + cls_id] = score;
  }
}

}  // namespace

template <>
bool CSCOp<float, HIPContext>::RunOnDevice() {
  const auto& M = Input(0);
  const auto& X = Input(1);
  const auto& Y = Input(2);
  const auto& R = Input(3);
  CAFFE_ENFORCE_EQ(M.dim(), 4);
  CAFFE_ENFORCE_EQ(X.dim(), 2);
  CAFFE_ENFORCE_EQ(Y.dim(), 2);
  CAFFE_ENFORCE_EQ(R.dim(), 2);
  CAFFE_ENFORCE_EQ(X.dim32(0), Y.dim32(0));
  CAFFE_ENFORCE_EQ(X.dim32(0), M.dim32(0));
  CAFFE_ENFORCE_EQ(X.dim32(1), Y.dim32(1));
  CAFFE_ENFORCE_EQ(X.dim32(1), M.dim32(1));
  CAFFE_ENFORCE_EQ(R.dim32(1), 5);

  const int batch_size = X.dim32(0);
  const int num_classes = X.dim32(1);
  const int num_rois = R.dim32(0);
  const int cpg_height = M.dim32(2);
  const int cpg_width = M.dim32(3);

  auto* W = Output(0);
  W->Resize(num_rois, num_classes);
  math::Set<float, HIPContext>(W->numel(), 1.f, W->mutable_data<float>(),
                                &context_);

  auto* PL = Output(1);
  PL->ResizeLike(X);
  PL->CopyFrom(X, false);
  context_.FinishDeviceComputation();

  auto* NL = Output(2);
  NL->ResizeLike(X);
  math::Set<float, HIPContext>(NL->numel(), 0.f, NL->mutable_data<float>(),
                                &context_);

  if (cur_iter_ >= max_iter_) {
    return true;
  }

  const int gpu_id = context_.device_id();
  int uuid;
  if (debug_info_) {
    srand(time(NULL));
    uuid = rand();
  }

  Tensor Xcpu = Tensor(X, caffe2::CPU);
  context_.FinishDeviceComputation();
  const float* Xcpudata = Xcpu.data<float>();

  Tensor Ycpu = Tensor(Y, caffe2::CPU);
  context_.FinishDeviceComputation();
  const float* Ycpudata = Ycpu.data<float>();

  for (int b = 0; b < batch_size; b++) {
    for (int c = 0; c < num_classes; c++) {
      int label_idx = b * num_classes + c;
      float label_value = Xcpudata[label_idx];
      float pred_value = Ycpudata[label_idx];
      if (debug_info_) {
        printf("uuid %d gpu %d b %d c %d: %.32f %.32f\n", uuid, gpu_id, b, c,
               label_value, pred_value);
      }
      if (label_value < 0.5) {
        continue;
      }
      // if (pred_value < tau_) {
      // continue;
      //}

      // Get CPG map
      Tensor m = Tensor(caffe2::CUDA);
      m.Resize(cpg_height, cpg_width);
      math::Abs<float, HIPContext>(
          m.numel(), M.data<float>() + cpg_height * cpg_width * label_idx,
          m.mutable_data<float>(), &context_);

      // Get max value
      Tensor mcpu = Tensor(m, caffe2::CPU);
      context_.FinishDeviceComputation();
      // float max_val = get_max<float>(mcpu.numel(), mcpu.data<float>());
      float max_val = 1.;
      if (debug_info_) {
        printf("uuid %d gpu %d max_val %.32f\n", uuid, gpu_id, max_val);
      }

      float im_mass = 0;
      float im_density = 0;
      // im_mass = get_sum<float>(mcpu.numel(), mcpu.data<float>());
      // im_density = 1.0 * im_mass / cpg_height / cpg_width;
      if (debug_info_) {
        printf("uuid %d gpu %d im_mass %.32f im_density %.32f\n", uuid, gpu_id,
               im_mass, im_density);
      }

      // Get Integral map
      Tensor icpu = Tensor(caffe2::CPU);
      icpu.ResizeLike(mcpu);
      binary_and_integral_cpu(mcpu.data<float>(), icpu.mutable_data<float>(),
                              cpg_height, cpg_width, max_val * fg_threshold_);
      // CAFFE_ENFORCE_EQ(icpu.data<float>()[cpg_height * cpg_width - 1],
      // im_mass);
      if (debug_info_) {
        printf("uuid %d gpu %d im_mass in icpu %.32f im_mass %.32f\n", uuid,
               gpu_id, icpu.data<float>()[cpg_height * cpg_width - 1], im_mass);
      }

      m.CopyFrom(icpu, false);
      context_.FinishDeviceComputation();

      // CSC Pooling
      CSCPool<float><<<CAFFE_GET_BLOCKS(num_rois), CAFFE_CUDA_NUM_THREADS, 0,
                       context_.hip_stream()>>>(
          num_rois, m.data<float>(), cpg_height, cpg_width, R.data<float>(),
          num_classes, c, im_density * density_threshold_,
          im_mass * mass_threshold_, area_sqrt_, context_scale_,
          W->mutable_data<float>());

      Tensor Wcpu = Tensor(*W, caffe2::CPU);
      context_.FinishDeviceComputation();
      // normalization max value to |1|
      float* Wcpudata = Wcpu.mutable_data<float>();
      float max_value = 0;
      float min_value = 0;
      for (int r = 0; r < num_rois; r++) {
        float value = Wcpudata[r * num_classes + c];
        if (value > max_value) {
          max_value = value;
        }
        if (value < min_value && value != kMIN_SCORE) {
          min_value = value;
        }
      }
      if (max_value > 0 && min_value < 0) {
        for (int r = 0; r < num_rois; r++) {
          float value = Wcpudata[r * num_classes + c];
          if (value == kMIN_SCORE) {
            value = -1;
          } else {
            value = value > 0 ? value / max_value : value / (-min_value);
          }
          // value = value > 0 ? value / max_value : -1;
          Wcpudata[r * num_classes + c] = value;
        }
      } else if (max_value > 0 && min_value == 0) {
        for (int r = 0; r < num_rois; r++) {
          float value = Wcpudata[r * num_classes + c];
          if (value == kMIN_SCORE) {
            value = -1;
          } else {
            value = value / max_value;
          }
          Wcpudata[r * num_classes + c] = value;
        }
      } else {
        for (int r = 0; r < num_rois; r++) {
          Wcpudata[r * num_classes + c] = 1.0;
        }
      }
      for (int r = 0; r < num_rois; r++) {
        Wcpudata[r * num_classes + c] =
            pred_value * Wcpudata[r * num_classes + c] + (1 - pred_value) * 1;
      }
      W->CopyFrom(Wcpu, &context_);
      context_.FinishDeviceComputation();

      if (debug_info_) {
        kernel_show_c<float>
            <<<CAFFE_GET_BLOCKS(1), 1, 0, context_.hip_stream()>>>(
                W->data<float>(), num_rois, num_classes, 1, 1, W->dim(),
                gpu_id, uuid, c);
      }
    }
  }

  cur_iter_++;

  return true;
}

REGISTER_HIP_OPERATOR(CSC, CSCOp<float, HIPContext>);

}  // namespace caffe2
