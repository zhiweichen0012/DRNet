#include <functional>

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/operator_fallback_gpu.h"
#include "instance_mining_op.h"

namespace caffe2 {

namespace {}  // namespace

REGISTER_HIP_OPERATOR(InstanceMining, GPUFallbackOp);

}  // namespace caffe2
